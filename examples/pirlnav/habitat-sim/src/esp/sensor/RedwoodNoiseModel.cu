#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include "RedwoodNoiseModel.cuh"

#include <algorithm>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

namespace {
int roundToNearestMultiple(int num, int base) {
  return std::round(static_cast<float>(num) / static_cast<float>(base)) * base;
}

const int MODEL_N_DIMS = 5;
const int MODEL_N_COLS = 80;

// Read about the noise model here: http://www.alexteichman.com/octo/clams/
// Original source code: http://redwood-data.org/indoor/data/simdepth.py
__device__ float undistort(const int _x,
                           const int _y,
                           const float z,
                           const float* __restrict__ model) {
  const int i2 = (z + 1) / 2;
  const int i1 = i2 - 1;
  const float a = (z - (i1 * 2.0f + 1.0f)) / 2.0f;
  const int x = _x / 8;
  const int y = _y / 6;

  const float f =
      (1.0f - a) *
          model[(y * MODEL_N_COLS + x) * MODEL_N_DIMS + min(max(i1, 0), 4)] +
      a * model[(y * MODEL_N_COLS + x) * MODEL_N_DIMS + min(i2, 4)];

  if (f < 1e-5f)
    return 0.0f;
  else
    return z / f;
}

__global__ void redwoodNoiseModelKernel(const float* __restrict__ depth,
                                        const int H,
                                        const int W,
                                        hiprandState_t* __restrict__ states,
                                        const float* __restrict__ model,
                                        const float noiseMultiplier,
                                        float* __restrict__ noisyDepth) {
  const int ID = blockIdx.x * blockDim.x + threadIdx.x;
  const int STRIDE = gridDim.x * blockDim.x;

  hiprandState_t hiprandState = states[ID];

  const float ymax = H - 1;
  const float xmax = W - 1;
  for (int idx = ID; idx < H * W; idx += STRIDE) {
    // Shuffle pixels
    const int y = min(max((idx / W) + hiprand_normal(&hiprandState) * 0.25f *
                                          noiseMultiplier,
                          0.0f),
                      ymax) +
                  0.5f;
    const int x = min(max((idx % W) + hiprand_normal(&hiprandState) * 0.25f *
                                          noiseMultiplier,
                          0.0f),
                      xmax) +
                  0.5f;

    // downsample
    const float d = depth[(y - y % 2) * W + x - x % 2];
    // If depth is greater than 10m, the sensor will just return a zero
    if (d >= 10.0f) {
      noisyDepth[idx] = 0.0f;
    } else {
      // Distortion
      // The noise model was originally made for a 640x480 sensor,
      // so re-map our arbitrarily sized sensor to that size!
      const float undistorted_d =
          undistort(static_cast<float>(x) / xmax * 639.0f + 0.5f,
                    static_cast<float>(y) / ymax * 479.0f + 0.5f, d, model);

      // quantization and high freq noise
      if (undistorted_d == 0.0f) {
        noisyDepth[idx] = 0.0f;
      } else {
        const float denom =
            round((35.130f / static_cast<double>(undistorted_d) +
                   hiprand_normal(&hiprandState) * 0.027778f * noiseMultiplier) *
                  8.0f);
        noisyDepth[idx] = denom > 1e-5 ? (35.130f * 8.0f / denom) : 0.0f;
      }
    }
  }

  states[ID] = hiprandState;
}

__global__ void curandStatesSetupKernel(hiprandState_t* states,
                                        int seed,
                                        int n) {
  const int ID = blockIdx.x * blockDim.x + threadIdx.x;
  if (ID < n) {
    hiprand_init(seed, ID + 1, 0, &states[ID]);
  }
}

}  // namespace

namespace esp {
namespace sensor {
namespace impl {

struct CurandStates {
  void alloc(const int nStates, const int maxThreadsPerBlock) {
    if (nStates > nStates_) {
      release();
      hipMalloc(&devStates, nStates * sizeof(hiprandState_t));
      const int nBlocks =
          std::ceil(static_cast<float>(nStates) / maxThreadsPerBlock);
      curandStatesSetupKernel<<<nBlocks, maxThreadsPerBlock>>>(devStates,
                                                               rand(), nStates);
      nStates_ = nStates;
    }
  }

  void release() {
    if (devStates != 0) {
      hipFree(devStates);
      devStates = 0;
      nStates_ = 0;
    }
  }

  ~CurandStates() { release(); }

  hiprandState_t* devStates = 0;

 private:
  int nStates_ = 0;
};

CurandStates* getCurandStates() {
  return new CurandStates();
}
void freeCurandStates(CurandStates* curandStates) {
  if (curandStates != 0)
    delete curandStates;
}

void simulateFromGPU(const int maxThreadsPerBlock,
                     const int warpSize,
                     const float* __restrict__ devDepth,
                     const int H,
                     const int W,
                     const float* __restrict__ devModel,
                     CurandStates* curandStates,
                     const float noiseMultiplier,
                     float* __restrict__ devNoisyDepth) {
  const int totalConcurrency = std::ceil(static_cast<float>(H * W) / 4.0f);
  const int nThreads =
      std::min(std::max(roundToNearestMultiple(totalConcurrency, warpSize), 1),
               maxThreadsPerBlock);
  const int nBlocks =
      std::ceil(static_cast<float>(totalConcurrency) / nThreads);

  curandStates->alloc(nBlocks * nThreads, maxThreadsPerBlock);
  redwoodNoiseModelKernel<<<nBlocks, nThreads>>>(
      devDepth, H, W, curandStates->devStates, devModel, noiseMultiplier,
      devNoisyDepth);
}

void simulateFromCPU(const int maxThreadsPerBlock,
                     const int warpSize,
                     const float* __restrict__ depth,
                     const int H,
                     const int W,
                     const float* __restrict__ devModel,
                     CurandStates* curandStates,
                     const float noiseMultiplier,
                     float* __restrict__ noisyDepth) {
  float *devDepth, *devNoisyDepth;
  hipMalloc(&devDepth, H * W * sizeof(float));
  hipMalloc(&devNoisyDepth, H * W * sizeof(float));

  hipMemcpy(devDepth, depth, H * W * sizeof(float), hipMemcpyHostToDevice);

  simulateFromGPU(maxThreadsPerBlock, warpSize, devDepth, H, W, devModel,
                  curandStates, noiseMultiplier, devNoisyDepth);

  hipMemcpy(noisyDepth, devNoisyDepth, H * W * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(devNoisyDepth);
  hipFree(devDepth);
}
}  // namespace impl
}  // namespace sensor
}  // namespace esp
